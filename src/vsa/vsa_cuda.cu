#include "hip/hip_runtime.h"
/**
 * Scotty3D - vsa_cuda.cu
 */
#include "vsa_cuda.h"

#define cudaCheckError(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
  {
    fprintf(stderr, "CUDA Error: %s at %s:%d\n",
            hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}
/****************************** Device Parameters *****************************/
#define THREADS_PER_BLOCK 512
#define NEAREST_PROXY_NUM 8

/*************************** Device Constant Memory ***************************/
__constant__ CudaVSAConfig cudaVsaConfig;

/********************************* Device Code ********************************/
__device__ __host__ __inline__ double distance2(double3 v1, double3 v2) {
  double dx = v1.x - v2.x, dy = v1.y - v2.y, dz = v1.z - v2.z;
  return dx*dx + dy*dy + dz*dz;
}
__device__ __host__ __inline__ double3 vecAdd(double3 v1, double3 v2) {
  return make_double3(v1.x+v2.x, v1.y+v2.y, v1.z+v2.z);
}

__device__ __host__ __inline__ double3 vecMul(double3 v1, double scaler) {
  return make_double3(v1.x*scaler, v1.y*scaler, v1.z*scaler);
}

__device__ double atomicMinDouble(double *addr, double val) {
  double old = *addr, assumed;
  if(old <= val) return old;
  do {
    assumed = old;
    old = atomicCAS((unsigned long long int*)addr, __double_as_longlong(assumed), __double_as_longlong(val));
  } while(old != assumed);
  return old;
}

#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ < 600
__device__ double atomicAdd(double* address, double val) {
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;
  do { assumed = old;
    old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
  } while (assumed != old);
  return __longlong_as_double(old);
}
#endif // #if __CUDA_ARCH__ < 600
#endif // #ifdef __CUDA_ARCH__

__global__ void kernelInitProxy(FaceCu *faces, ProxyCu *proxies) {
  // TODO: k-means++ initialization or some other smart initialization

  // a naive fixed initialization
  size_t offset = cudaVsaConfig.numFaces / cudaVsaConfig.numProxies;
  FaceIndex index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= cudaVsaConfig.numFaces) return;
  if (index % offset == 0) {
    ProxyLabel proxyLabel = index / offset;
    faces[index].label = proxyLabel;
    proxies[proxyLabel].seed      = index;
    proxies[proxyLabel].normal    = faces[index].normal;
    proxies[proxyLabel].centroid  = faces[index].centroid;
  }
}

__global__ void kernelFlood(FaceCu *faces, ProxyCu *proxies) {

  FaceIndex index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= cudaVsaConfig.numFaces) return;
  double nearestProxyDist[NEAREST_PROXY_NUM];
  ProxyLabel nearestProxyLabel[NEAREST_PROXY_NUM];
  double maxDist = 0;
  size_t maxDistIdx = 0;

  // TODO use shared memeory for proxy centroids and normals
  // Find the nearest <NEAREST_PROXY_NUM> proxies to this face
  for (ProxyLabel l = 0; l < cudaVsaConfig.numProxies; l++) {
    double dist2 = distance2(faces[index].centroid, proxies[l].centroid);
    if (l < NEAREST_PROXY_NUM) {
      nearestProxyDist[l] = dist2;
      nearestProxyLabel[l] = l;
      if (dist2 > maxDist) {
        maxDist = dist2;
        maxDistIdx = (size_t)l;
      }
    } else { // l >= NEAREST_PROXY_NUM
      // TODO consider reduce conditional branches
      // replace proxy with largest distance with the current proxy
      if (dist2 < maxDist) {
        nearestProxyDist[maxDistIdx] = dist2;
        nearestProxyLabel[maxDistIdx] = l;
        maxDist = 0;
        for (size_t i = 0; i < NEAREST_PROXY_NUM; i++) {
          if (nearestProxyDist[i] > maxDist) {
            maxDist = nearestProxyDist[i];
            maxDistIdx = i;
          }
        }
      }
    }
  }

  // Among those nearest proxies find one that has the minimum distortion
  // error and assign its label to this host
  ProxyLabel labelToAssign = -1;
  double minError = 4.0; // this is the maximum distance2 of two normalized vector
  for (size_t i = 0; i < NEAREST_PROXY_NUM; i++) {
    double error = distance2(faces[index].normal,
                             proxies[nearestProxyLabel[i]].normal);
    if (error < minError) {
      minError = error;
      labelToAssign = nearestProxyLabel[i];
    }
  }

  faces[index].label = labelToAssign;
}

__global__ void kernelAddUpProxy(FaceCu *faces, ProxyCu *proxies) {
  FaceIndex index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= cudaVsaConfig.numFaces) return;
  ProxyLabel l = faces[index].label;
  double3 areaNormal = vecMul(faces[index].normal, faces[index].area);
  double3 areaCentroid = vecMul(faces[index].centroid, faces[index].area);

  // update total area of proxy
  atomicAdd(&(proxies[l].totalArea), faces[index].area);

  // update normal of proxy
  atomicAdd(&(proxies[l].normal.x), areaNormal.x);
  atomicAdd(&(proxies[l].normal.y), areaNormal.y);
  atomicAdd(&(proxies[l].normal.z), areaNormal.z);

  // update centroid of proxy
  atomicAdd(&(proxies[l].centroid.x), areaCentroid.x);
  atomicAdd(&(proxies[l].centroid.y), areaCentroid.y);
  atomicAdd(&(proxies[l].centroid.z), areaCentroid.z);
}

__global__ void kernelUpdateProxy(ProxyCu *proxies) {
  ProxyLabel label = blockIdx.x * blockDim.x + threadIdx.x;
  if (label >= cudaVsaConfig.numProxies) return;
  proxies[label].centroid = vecMul(proxies[label].centroid, 1.0/proxies[label].totalArea);
  proxies[label].normal = vecMul(proxies[label].normal, 1.0/proxies[label].totalArea);
}

__global__ void kernelUpdateSeedFace(FaceCu *faces, ProxyCu *proxies, double *lowestNormalDiff) {
  FaceIndex index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < cudaVsaConfig.numProxies) {
    lowestNormalDiff[index] = 4.0;
  }
  __syncthreads();

  if (index >= cudaVsaConfig.numFaces || faces[index].isBoundary) return;
  // update seed face for final round of flooding
  ProxyLabel label = faces[index].label;
  double diff = distance2(faces[index].normal, proxies[label].normal);

  // TODO Fix race condition here
  double oldDiff = atomicMinDouble(&lowestNormalDiff[label], diff);
  if (oldDiff != diff) {
    proxies[label].seed = index;
  }
  // clear proxy label of a face
  faces[index].label = -1;
}

/********************************** Host Code *********************************/
CudaVSAPartitioner::CudaVSAPartitioner(std::vector<FaceCu> *hostFaceCu,
                                       std::vector<ProxyCu> *hostProxyCu) {
  this->hostFaceCu = hostFaceCu;
  this->hostProxyCu = hostProxyCu;
  this->vsaConfig.numFaces = hostFaceCu->size();
  this->vsaConfig.numProxies = hostProxyCu->size();
  deviceFaceCu = NULL;
  deviceProxyCu = NULL;
  setup();
}

CudaVSAPartitioner::~CudaVSAPartitioner() {
  if(deviceProxyCu) {
    hipFree(deviceProxyCu);
    hipFree(deviceFaceCu);
    hipFree(deviceLowestNormalDiff);
    cudaCheckError(hipDeviceSynchronize());
//    hipFree(deviceFaceNearestProxy);
  }
}

void CudaVSAPartitioner::setup() {

  int deviceCount = 0;
  std::string name;
  hipError_t err = hipGetDeviceCount(&deviceCount);

  printf("---------------------------------------------------------\n");
  printf("Initializing CUDA for VSA Partitioner\n");
  printf("Found %d CUDA devices\n", deviceCount);

  for (int i = 0; i < deviceCount; i++) {
    hipDeviceProp_t deviceProps;
    hipGetDeviceProperties(&deviceProps, i);
    name = deviceProps.name;

    printf("Device %d: %s\n", i, deviceProps.name);
    printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
    printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
    printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
  }
  printf("---------------------------------------------------------\n");

  // allocate device memory
  double startTime, endTime;
  startTime = CycleTimer::currentSeconds();
  cudaCheckError(hipMalloc(&deviceFaceCu, sizeof(FaceCu) * vsaConfig.numFaces));
  cudaCheckError(hipMalloc(&deviceProxyCu, sizeof(ProxyCu) * vsaConfig.numProxies));
  cudaCheckError(hipMalloc(&deviceLowestNormalDiff, sizeof(double) * vsaConfig.numProxies));
  cudaCheckError(hipDeviceSynchronize());
  endTime = CycleTimer::currentSeconds();
  fprintf(stdout, "[VSA CUDA] hipMalloc()        (%.4f sec)\n", endTime - startTime);
//  cudaCheckError(hipMalloc(&deviceFaceNearestProxy, sizeof(ProxyLabel) * vsaConfig.numFaces));
}

void CudaVSAPartitioner::partition(size_t numIterations) {
  // TODO allocate result proxy mapping data structure which is essentially map of (face -> proxy label)
  double startTime, endTime;
  // Copy data from host to device
  startTime = CycleTimer::currentSeconds();
  cudaCheckError(hipMemcpy(deviceFaceCu, &(*hostFaceCu)[0],
                            sizeof(FaceCu) * hostFaceCu->size(), hipMemcpyHostToDevice));
  cudaCheckError(hipMemcpy(deviceProxyCu, &(*hostProxyCu)[0],
                            sizeof(ProxyCu) * hostProxyCu->size(), hipMemcpyHostToDevice));
  cudaCheckError(hipMemcpyToSymbol(HIP_SYMBOL(cudaVsaConfig), &vsaConfig, sizeof(CudaVSAConfig)));
  cudaCheckError(hipDeviceSynchronize());
  endTime = CycleTimer::currentSeconds();
  fprintf(stdout, "[VSA CUDA] Host to Device I/O  (%.4f sec)\n", endTime - startTime);

  // Actual Compute
  startTime = CycleTimer::currentSeconds();
  initProxy();
  endTime = CycleTimer::currentSeconds();
  fprintf(stdout, "[VSA CUDA] Initialization Time (%.4f sec)\n", endTime - startTime);
  double flooding_time = 0.0, fitting_time = 0.0, sequential_flooding_time = 0.0;

  startTime = CycleTimer::currentSeconds();
  flood();
  endTime = CycleTimer::currentSeconds();
  flooding_time += endTime - startTime;

  // Lloyd Iterations
  for (size_t i = 1; i < numIterations; i++) {
    startTime = CycleTimer::currentSeconds();
    fitProxy(false);
    endTime = CycleTimer::currentSeconds();
    fitting_time += endTime - startTime;

    startTime = CycleTimer::currentSeconds();
    flood();
    endTime = CycleTimer::currentSeconds();
    flooding_time += endTime - startTime;
  }

  // fit proxy with seed face update
  startTime = CycleTimer::currentSeconds();
  fitProxy(true);
  endTime = CycleTimer::currentSeconds();
  fitting_time += endTime - startTime;


  fprintf(stdout, "[VSA CUDA] Flooding Time       (%.4f sec)\n", flooding_time);
  fprintf(stdout, "[VSA CUDA] Proxy fitting Time  (%.4f sec)\n", fitting_time);

  // Copy data back to host
  startTime = CycleTimer::currentSeconds();
  cudaCheckError(hipMemcpy(&(*hostFaceCu)[0], deviceFaceCu,
                            sizeof(FaceCu) * vsaConfig.numFaces, hipMemcpyDeviceToHost));
  cudaCheckError(hipMemcpy(&(*hostProxyCu)[0], deviceProxyCu,
                            sizeof(ProxyCu) * vsaConfig.numProxies, hipMemcpyDeviceToHost));
  cudaCheckError(hipDeviceSynchronize());
  endTime = CycleTimer::currentSeconds();
  fprintf(stdout, "[VSA CUDA] Device to Host I/O  (%.4f sec)\n", endTime - startTime);

  startTime = CycleTimer::currentSeconds();
  sequentialFlood();
  endTime = CycleTimer::currentSeconds();
  sequential_flooding_time += endTime - startTime;
  fprintf(stdout, "[VSA CUDA] Seq Flooding Time   (%.4f sec)\n", sequential_flooding_time);
}

/** Initialize Proxy Randomly */
void CudaVSAPartitioner::initProxy() {
  const size_t blocks = (vsaConfig.numFaces + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  kernelInitProxy<<<blocks, THREADS_PER_BLOCK>>>(deviceFaceCu, deviceProxyCu);
  cudaCheckError(hipDeviceSynchronize());
}
/**
 * A data parallel approach of distortion minimizing flooding:
 *
 * 1. We first compute the euclidean distance (as approximation to geodesic distance)
 *    of each face to each proxy centroid.
 * 2. Pick the nearest (maybe 4) proxies of each face and compute the differences of
 *    their normals. Assign proxy label of least normal diff to face.
 * 3. (Maybe we can omit this step) Clean up disconnected clusters with same label
 *    using BFS, keep the cluster with largest total area, unlabel the rest of faces
 * 4. Re-flood using the sequential method to ensure connectivity
 *
 * Inspired by: Fan, Fengtao, et al. "Mesh clustering by approximating centroidal Voronoi tessellation."
 */
void CudaVSAPartitioner::flood() {
  const size_t blocks = (vsaConfig.numFaces + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  kernelFlood<<<blocks, THREADS_PER_BLOCK>>>(deviceFaceCu, deviceProxyCu);

  cudaCheckError(hipDeviceSynchronize());
}
/** Proxy Fitting using L_2,1 Error Metric */
void CudaVSAPartitioner::fitProxy(bool updateSeedFace) {
  // clear old proxy values
  hipMemset(deviceProxyCu, 0, sizeof(ProxyCu) * vsaConfig.numProxies);
  cudaCheckError(hipDeviceSynchronize());
  // add up related values per proxy
  const size_t faceBlocks = (vsaConfig.numFaces + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  kernelAddUpProxy<<<faceBlocks, THREADS_PER_BLOCK>>>(deviceFaceCu, deviceProxyCu);
  cudaCheckError(hipDeviceSynchronize());
  // update normal and seed of all proxies
  const size_t proxyBlocks = (vsaConfig.numProxies + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  kernelUpdateProxy<<<proxyBlocks, THREADS_PER_BLOCK>>>(deviceProxyCu);
  cudaCheckError(hipDeviceSynchronize());
  // We pick the face with the lowest difference of normal as the new seed
  if (updateSeedFace) {
    kernelUpdateSeedFace <<<faceBlocks, THREADS_PER_BLOCK>>>(deviceFaceCu, deviceProxyCu, deviceLowestNormalDiff);
    cudaCheckError(hipDeviceSynchronize());
  }
}

void CudaVSAPartitioner::sequentialFlood() {
  // Global priority queue of faces
  std::multiset<MetricFaceCu, MetricFaceComp> faceQueue;

  // For each proxy, first add its seed face to priority queue
  for (ProxyLabel label = 0; label < vsaConfig.numProxies; label++) {
    ProxyCu proxy = (*hostProxyCu)[label];
    FaceIndex seedFace = proxy.seed;

    MetricFaceCu m;
    m.faceIndex = seedFace;
    m.distance = 0.0;
    m.possibleLabel = label;

    faceQueue.insert(m);
  }

  // For each face in the priority queue, perform flooding by adding its adjacent faces into the priority queue
  for (FaceQueueIter queueIter = faceQueue.begin(); queueIter != faceQueue.end(); ) {
    FaceIndex face = queueIter->faceIndex;
    FaceCu *facePtr = &(*hostFaceCu)[face];
    if (facePtr->label == -1) { // this means the face hasn't been labeled before
      // label this face, because it is supposed to have the least distortion error
      facePtr->label = queueIter->possibleLabel;
      // add all unlabeled adjacent faces to the priority queue
      for (int i = 0; i < 3; i++) {
        FaceIndex neighborIndex = facePtr->neighbors[i];
        if (neighborIndex < 0) continue;
        FaceCu *neighborPtr = &(*hostFaceCu)[neighborIndex];
        if (!neighborPtr->isBoundary && neighborPtr->label == -1) {
          MetricFaceCu m;
          m.faceIndex = neighborIndex;
          m.distance = neighborPtr->area *
                       distance2(neighborPtr->normal, (*hostProxyCu)[queueIter->possibleLabel].normal);
          m.possibleLabel = queueIter->possibleLabel;
          faceQueue.insert(m);
        }
      }
    }
    // pop the top of priority queue
    faceQueue.erase(queueIter);
    queueIter = faceQueue.begin();
  }
}